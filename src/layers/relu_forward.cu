#include "hip/hip_runtime.h"
#include "relu_forward.cuh"

__global__ void relu_forward_kernel(
    float* input,
    int N, int C, int H, int W
)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int size = N * C * H * W;

    if (index < size)
    {
        input[index] = fmaxf(0.0f, input[index]);
    }
}